#include "hip/hip_runtime.h"
// matrix_add.cu
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <iostream>
using namespace std;

__global__ void add_kernel(const float* a, const float* b, float* c, unsigned int width, unsigned int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row*width+col;
    if ((row<height) && (col<width)) {
        c[index] = a[index] + b[index];
    }
}

void add_cuda(const torch::Tensor& a, const torch::Tensor& b, torch::Tensor& c) {
    unsigned int height = a.size(0);
    unsigned int width = a.size(1);
    dim3 dimBlock(32,32);
    cout << width << height << dimBlock.x << dimBlock.y << ceil(width/dimBlock.x) << ceil(height/dimBlock.y) << endl;
    dim3 dimGrid(ceil((float)width/dimBlock.x),ceil((float)height/dimBlock.y));

    add_kernel<<<dimGrid, dimBlock>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), width, height);
}