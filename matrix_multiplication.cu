#include "hip/hip_runtime.h"
// matrix_multiplication.cu
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <iostream>
using namespace std; 

__global__ void mul_kernel(const float* a, const float* b, float* c, unsigned int width, unsigned int height, unsigned int k) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row>=height || col>=width) {
        return;
    }

    float tmp = 0;
    for (int i=0; i<k; i++) {
        tmp += a[row*width+i] * b[width*i+col];
    }
    c[row*width+col] = tmp;
}

#define TILE_WIDTH 16
__global__ void mul_kernel_tiles(const float* a, const float* b, float* c, unsigned int width, unsigned int height, unsigned int k) {
    __shared__ float a_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float b_tile[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col = blockIdx.x * TILE_WIDTH + tx;
    int row = blockIdx.y * TILE_WIDTH + ty;

    float tmp = 0;
    for (int t = 0; t<ceil((float)k/TILE_WIDTH); ++t) {
        int a_id = row*k+t*TILE_WIDTH+tx;
        if (t*TILE_WIDTH+tx < k && row<height) {
            a_tile[ty][tx] = a[a_id];
        } else {
            a_tile[ty][tx] = 0;
        }

        int b_id = ty*width+t*TILE_WIDTH*width+col;
        if (ty+t*TILE_WIDTH < k && col<width) {
            b_tile[ty][tx] = b[b_id];
        } else {
            b_tile[ty][tx] = 0;
        }
        __syncthreads();

        // calclate dot product
        for (int i=0; i<TILE_WIDTH; ++i) {
            tmp += a_tile[ty][i] * b_tile[i][tx];
        }
        __syncthreads();
    }
    if (row<height && col<width)
        c[row*width+col] = tmp;
}

void mul_cuda(const torch::Tensor& a, const torch::Tensor& b, torch::Tensor& c) {
    unsigned int height = a.size(0);
    unsigned int k = a.size(1);
    unsigned int width = b.size(1);
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH);
    dim3 dimGrid(ceil((float)width/dimBlock.x),ceil((float)height/dimBlock.y));

    cout << width << height << width << k << endl;
    mul_kernel_tiles<<<dimGrid, dimBlock>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), width, height, k);
}